
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CHECK_CUDA(func)                                                  \
  {                                                                       \
    hipError_t status = (func);                                          \
    if (status != hipSuccess) {                                          \
      printf("CUDA API failed at file %s, line %d with error: %s (%d)\n", \
             __FILE__, __LINE__, hipGetErrorString(status), status);     \
      exit(EXIT_FAILURE);                                                 \
    }                                                                     \
  }

__global__ void foo() {
  int temp[33];
}

int main(int argc, char** argv) {
  auto device = 0;
  hipDeviceProp_t deviceProp;
  CHECK_CUDA(hipGetDeviceProperties(&deviceProp, device));

  // Setup grid and block properties
  auto numBlocksPerSm = 0;
  auto numThreadsPerBlock = 0;
  int shmemPerBlock = 0;  // bytes

  // Use the max number of threads per block to maximize parallelism over
  // shmem
  auto target_occupancy = 2;
  numThreadsPerBlock = deviceProp.maxThreadsPerBlock / target_occupancy;
  shmemPerBlock =
      (deviceProp.sharedMemPerBlockOptin - 1024 * target_occupancy) /
      target_occupancy;

  int carveout = 100;
  CHECK_CUDA(hipFuncSetAttribute(
      reinterpret_cast<const void*>(foo), hipFuncAttributePreferredSharedMemoryCarveout, carveout));

  std::cout << "Threads Per Block: " << numThreadsPerBlock << std::endl;
  std::cout << "Shmem Per Block (bytes): " << shmemPerBlock << std::endl;

  CHECK_CUDA(hipFuncSetAttribute(
      reinterpret_cast<const void*>(foo), hipFuncAttributeMaxDynamicSharedMemorySize, shmemPerBlock));

  // Need to know the max occupancy to determine how many blocks to launch
  // for the cooperative kernel. All blocks must be resident on SMs
  CHECK_CUDA(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &numBlocksPerSm, foo, numThreadsPerBlock, shmemPerBlock))

  // See how many registers the kernel uses
  hipFuncAttributes attr;
  CHECK_CUDA(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(foo)));

  std::cout << "Registers: " << attr.numRegs << std::endl;

  std::cout << "Max Active Blocks Per SM: " << numBlocksPerSm << std::endl;

  foo<<<1024, numThreadsPerBlock>>>();

  CHECK_CUDA(hipDeviceSynchronize());
}