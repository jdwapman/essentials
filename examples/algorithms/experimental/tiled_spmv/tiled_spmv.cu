#include <gunrock/algorithms/algorithms.hxx>
#include <hip/hip_runtime_api.h>
#include <cxxopts.hpp>

#include "spmv_cpu.hxx"
#include "spmv_cusparse.cuh"
#include "spmv_cub.cuh"
#include "spmv_moderngpu.cuh"
#include "test_tiled.h"
#include "spmv_utils.cuh"
#include "launch_params.cuh"

using namespace gunrock;
// using namespace experimental;
using namespace memory;

enum SPMV_t { MGPU, CUB, CUSPARSE, TILED };
enum LB_t {
  THREAD_PER_ROW,
  WARP_PER_ROW,
  BLOCK_PER_ROW,
  MERGE_PATH,
  NONZERO_SPLIT,
  TWC
};

template <typename csr_t, typename vector_t>
double test_spmv(SPMV_t spmv_impl,
                 csr_t& sparse_matrix,
                 vector_t& d_input,
                 vector_t& d_output,
                 bool cpu_verify,
                 bool debug,
                 bool ampere_cache) {
  // Reset the output vector
  thrust::fill(d_output.begin(), d_output.end(), 0);

  // if (ampere_cache) {
  //   stream = setup_ampere_cache(stream);
  // } else {
  // }

  double elapsed_time = 0;

  //   Run on appropriate GPU implementation
  if (spmv_impl == MGPU) {
    printf("=== RUNNING MODERNGPU SPMV ===\n");
    elapsed_time = spmv_mgpu(sparse_matrix, d_input, d_output);
  } else if (spmv_impl == CUB) {
    printf("=== RUNNING CUB SPMV ===\n");
    elapsed_time = spmv_cub( sparse_matrix, d_input, d_output);
  } else if (spmv_impl == CUSPARSE) {
    printf("=== RUNNING CUSPARSE SPMV ===\n");
    elapsed_time = spmv_cusparse(sparse_matrix, d_input, d_output);
  } else if (spmv_impl == TILED) {
    printf("=== RUNNING TILED SPMV ===\n");
    elapsed_time = spmv_tiled(sparse_matrix, d_input, d_output);
  } else {
    std::cout << "Unsupported SPMV implementation" << std::endl;
  }


  if (debug)
    printf("GPU finished in %lf ms\n", elapsed_time);

  //   Copy argss to CPU
  if (cpu_verify) {
    thrust::host_vector<float> h_output = d_output;
    thrust::host_vector<float> h_input = d_input;

    // Run on CPU
    thrust::host_vector<float> cpu_ref(sparse_matrix.number_of_rows);
    cpu_spmv(sparse_matrix, h_input, cpu_ref);

    if (debug) {
      display(d_input, "d_input");
      display(d_output, "d_output");
      display(cpu_ref, "cpu_ref");
    }

    // Validate
    int num_errors = check_spmv(cpu_ref, h_output);

    // Print the number of errors
    if (debug)
      printf("Errors: %d\n", num_errors);

    if (!num_errors) {
      if (debug)
        std::cout << "Validation Successful" << std::endl;
      return elapsed_time;
    } else {
      if (debug)
        std::cout << "Validation Failed" << std::endl;
      return -1;
    }
  }

  return elapsed_time;
}

void test_spmv(int num_arguments, char** argument_array) {
  cxxopts::Options options(argument_array[0], "Tiled SPMV");

  options.add_options()  // Allows to add options.
      ("b,bin", "CSR binary file",
       cxxopts::value<std::string>())  // CSR
      ("m,market", "Matrix-market format file",
       cxxopts::value<std::string>())  // Market
      ("c,cache", "Use Ampere cache pinning",
       cxxopts::value<bool>()->default_value("false"))  // Market
      ("g,gpu", "GPU to run on",
       cxxopts::value<int>()->default_value("0"))  // GPU
      ("v,verbose", "Verbose output",
       cxxopts::value<bool>()->default_value("false"))  // Verbose (not used)
      ("h,help", "Print help");                         // Help

  auto args = options.parse(num_arguments, argument_array);

  if (args.count("help") ||
      (args.count("market") == 0 && args.count("csr") == 0)) {
    std::cout << options.help({""}) << std::endl;
    std::exit(0);
  }

  // TODO set the GPU appropriately

  // Get the number of GPUs in the system
  int num_gpus = 0;
  hipGetDeviceCount(&num_gpus);
  std::cout << "Number of GPUs: " << num_gpus << std::endl;

  // Check if the GPU is valid
  if (args["gpu"].as<int>() >= num_gpus) {
    std::cout << "Invalid GPU" << std::endl;
    return;
  }

  printf("Using GPU %d\n", args["gpu"].as<int>());
  CHECK_CUDA(hipSetDevice(args["gpu"].as<int>()));

  std::string filename = "";
  if (args.count("market") == 1) {
    filename = args["market"].as<std::string>();
    if (util::is_market(filename)) {
    } else {
      std::cout << options.help({""}) << std::endl;
      std::exit(0);
    }
  } else if (args.count("csr") == 1) {
    filename = args["csr"].as<std::string>();
    if (util::is_binary_csr(filename)) {
    } else {
      std::cout << options.help({""}) << std::endl;
      std::exit(0);
    }
  } else {
    std::cout << options.help({""}) << std::endl;
    std::exit(0);
  }

  // --
  // Define types

  using row_t = int;
  using edge_t = int;
  using nonzero_t = float;

  using csr_t = format::csr_t<memory_space_t::device, row_t, edge_t, nonzero_t>;

  // --
  // IO

  csr_t csr;
  if (util::is_market(filename)) {
    io::matrix_market_t<row_t, edge_t, nonzero_t> mm;
    csr.from_coo(mm.load(filename));
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  // Print the GPU stats
  print_gpu_stats();

  // Print the matrix stats
  printf("Matrix: %s\n", filename.c_str());
  printf("- Rows: %d\n", csr.number_of_rows);
  printf("- Nonzeros: %d\n", csr.number_of_nonzeros);

  thrust::host_vector<nonzero_t> x_host(csr.number_of_columns);

  srand(0);
  for (size_t idx = 0; idx < x_host.size(); idx++)
    x_host[idx] = rand() % 64;

  thrust::device_vector<nonzero_t> x_device = x_host;
  thrust::device_vector<nonzero_t> y_device(csr.number_of_rows);

  // --
  // Run the algorithm

  bool cpu_verify = true;
  bool debug = true;
  bool ampere_cache = args["cache"].as<bool>();

  // NOTE: Can't seem to pass the args into the function here
  // double elapsed_cusparse = test_spmv(CUSPARSE, csr, x_device, y_device,
  //                                     cpu_verify, debug, ampere_cache);

  double elapsed_cub =
      test_spmv(CUB, csr, x_device, y_device, cpu_verify, debug, ampere_cache);

  // double elapsed_mgpu = test_spmv(MGPU, csr, x_device, y_device,
  //                                 cpu_verify, debug, ampere_cache);

  double elapsed_tiled = test_spmv(TILED, csr, x_device, y_device, cpu_verify,
                                   debug, ampere_cache);

  // printf("%s,%d,%d,%d,%f,%f,%f,%f\n", filename.c_str(), csr.number_of_rows,
  //        csr.number_of_columns, csr.number_of_nonzeros, elapsed_cusparse,
  //        elapsed_cub, elapsed_mgpu, elapsed_tiled);

  /* ========== RESET THE GPU ========== */

  // if (deviceProp.major >= 8)
  // {
  //   // Setting the window size to 0 disable it
  //   stream_attribute.accessPolicyWindow.num_bytes = 0;

  //   // Overwrite the access policy attribute to a CUDA Stream
  //   CHECK_CUDA(hipStreamSetAttribute(
  //       stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute));

  //   // Remove any persistent lines in L2
  //   CHECK_CUDA(cudaCtxResetPersistingL2Cache());
  // }
}

int main(int argc, char** argv) {
  test_spmv(argc, argv);
  return EXIT_SUCCESS;
}
