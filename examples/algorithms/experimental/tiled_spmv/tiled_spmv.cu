#include "hip/hip_runtime.h"
#include <gunrock/algorithms/algorithms.hxx>
#include <hip/hip_runtime_api.h>
#include <cxxopts.hpp>
#include <iostream>
#include <ctime>

#include "spmv_cpu.hxx"
#include "spmv_cusparse.cuh"
#include "spmv_cub.cuh"
#include "spmv_moderngpu.cuh"
#include "spmv_tiled.cuh"
#include "spmv_utils.cuh"
#include "launch_params.cuh"
#include "log.h"
#include <gunrock/algorithms/spmv.hxx>
#include <nlohmann/json.hpp>
#include <typeinfo>
#include <unistd.h>
#include <fstream>

// for convenience
using json = nlohmann::json;

// using namespace experimental;
// using namespace memory;

enum SPMV_t { MGPU, CUB, CUSPARSE, GUNROCK, TILED };
enum LB_t {
  THREAD_PER_ROW,
  WARP_PER_ROW,
  BLOCK_PER_ROW,
  MERGE_PATH,
  NONZERO_SPLIT,
  TWC
};

auto to_string(SPMV_t t) {
  switch (t) {
    case MGPU:
      return "mgpu";
    case CUB:
      return "cub";
    case CUSPARSE:
      return "cusparse";
    case GUNROCK:
      return "gunrock";
    case TILED:
      return "tiled";
    default:
      return "unknown";
  }
}

template <typename csr_t, typename vector_t, typename args_t>
double test_spmv(SPMV_t spmv_impl,
                 csr_t& sparse_matrix,
                 vector_t& d_input,
                 vector_t& d_output,
                 args_t pargs,
                 json& _results) {
  // Reset the output vector
  thrust::fill(d_output.begin(), d_output.end(), 0);

  // auto device = pargs["gpu"].template as<int>();

  hipStream_t stream;
  if (pargs.count("pin")) {
    stream = setup_ampere_cache(d_input, _results);
  } else {
    CHECK_CUDA(hipStreamCreate(&stream));
  }

  double elapsed_time = 0;

  //   Run on appropriate GPU implementation
  if (spmv_impl == MGPU) {
    printf("=== RUNNING MODERNGPU SPMV ===\n");
    elapsed_time = spmv_mgpu(stream, sparse_matrix, d_input, d_output, pargs);
  } else if (spmv_impl == CUB) {
    printf("=== RUNNING CUB SPMV ===\n");
    elapsed_time = spmv_cub(stream, sparse_matrix, d_input, d_output, pargs);
  } else if (spmv_impl == CUSPARSE) {
    printf("=== RUNNING CUSPARSE SPMV ===\n");
    elapsed_time =
        spmv_cusparse(stream, sparse_matrix, d_input, d_output, pargs);
  } else if (spmv_impl == TILED) {
    printf("=== RUNNING TILED SPMV ===\n");
    elapsed_time =
        spmv_tiled(stream, sparse_matrix, d_input, d_output, pargs, _results);
  } else if (spmv_impl == GUNROCK) {
    printf("=== RUNNING GUNROCK SPMV ===\n");
    auto G = gunrock::graph::build::from_csr<gunrock::memory_space_t::device,
                                             gunrock::graph::view_t::csr>(
        sparse_matrix.number_of_rows, sparse_matrix.number_of_columns,
        sparse_matrix.number_of_nonzeros,
        sparse_matrix.row_offsets.data().get(),
        sparse_matrix.column_indices.data().get(),
        sparse_matrix.nonzero_values.data().get());

    // Create the context
    std::shared_ptr<gunrock::gcuda::multi_context_t> context =
        std::shared_ptr<gunrock::gcuda::multi_context_t>(
            new gunrock::gcuda::multi_context_t(0, stream));
    elapsed_time = gunrock::spmv::run(G, d_input.data().get(),
                                      d_output.data().get(), context);
  } else {
    std::cout << "Unsupported SPMV implementation" << std::endl;
  }

  if (pargs.count("pin")) {
    reset_ampere_cache(stream);
  }

  if (pargs.count("verbose"))
    printf("GPU finished in %lf ms\n", elapsed_time);

  // Copy data to CPU
  if (pargs.count("cpu")) {
    thrust::host_vector<float> h_output = d_output;
    thrust::host_vector<float> h_input = d_input;

    // Run on CPU
    thrust::host_vector<float> cpu_ref(sparse_matrix.number_of_rows);
    cpu_spmv(sparse_matrix, h_input, cpu_ref);

    if (pargs.count("verbose")) {
      display(d_input, "d_input", 40);
      display(d_output, "d_output", 40);
      display(cpu_ref, "cpu_ref", 40);
    }

    // Validate
    int num_errors = check_spmv(cpu_ref, h_output, pargs);

    _results["num_errors"][to_string(spmv_impl)] = num_errors;

    // Print the number of errors
    if (pargs.count("verbose"))
      printf("Errors: %d\n", num_errors);

    if (!num_errors) {
      if (pargs.count("verbose"))
        std::cout << "Validation Successful" << std::endl;
      return elapsed_time;
    } else {
      if (pargs.count("verbose"))
        std::cout << "Validation Failed" << std::endl;
      return -1;
    }
  }

  return elapsed_time;
}

void test_spmv(int num_arguments, char** argument_array) {
  cxxopts::Options options(argument_array[0], "Tiled SPMV");

  options.add_options()  // Allows to add options.
      ("b,bin", "CSR binary file",
       cxxopts::value<std::string>())  // CSR
      ("m,market", "Matrix-market format file",
       cxxopts::value<std::string>())  // Market
      ("j,jsonfile", "json output filename. Can also be stdout",
       cxxopts::value<std::string>()->default_value("results.json"))  // JSON
      ("c,cpu", "Run a CPU comparison",
       cxxopts::value<bool>()->default_value("false"))  // CPU
      ("cub", "Run CUB SPMV",
       cxxopts::value<bool>()->default_value("false"))  // CUB
      ("mgpu", "Run ModernGPU SPMV",
       cxxopts::value<bool>()->default_value("false"))  // MGPU
      ("cusparse", "Run cuSparse SPMV",
       cxxopts::value<bool>()->default_value("false"))  // cuSparse
      ("gunrock", "Run Gunrock SPMV",
       cxxopts::value<bool>()->default_value("false"))  // Gunrock
      ("tiled", "Run Tiled SPMV",
       cxxopts::value<bool>()->default_value("false"))  // Tiled
      ("p,pin", "Use Ampere L2 cache pinning",
       cxxopts::value<bool>()->default_value("false"))  // Ampere L2
      ("d,device", "Device to run on",
       cxxopts::value<int>()->default_value("0"))  // GPU
      ("v,verbose", "Verbose output",
       cxxopts::value<bool>()->default_value("false"))  // Verbose
      ("h,help", "Print help");                         // Help

  json results;

  // Save command line options to the json

  auto args = options.parse(num_arguments, argument_array);

  log_cmd_args(results, args);

  // Save the current date and time to the json. But strip the newline
  time_t now = time(0);
  char* dt = ctime(&now);
  // Strip the newline from dt
  dt[strlen(dt) - 1] = '\0';
  results["time_local"] = dt;

  // Strip the newline from utc_time
  auto utc_time = asctime(gmtime(&now));
  utc_time[strlen(utc_time) - 1] = '\0';
  results["time_utc"] = utc_time;

  // Save the hostname
  char hostname[1024];
  gethostname(hostname, 1024);
  results["hostname"] = hostname;

  // Save the current git commit

  if (args.count("help") ||
      (args.count("market") == 0 && args.count("bin") == 0)) {
    std::cout << options.help({""}) << std::endl;
    std::exit(0);
  }

  // Get the number of GPUs in the system
  int num_gpus = 0;
  hipGetDeviceCount(&num_gpus);
  std::cout << "Number of GPUs: " << num_gpus << std::endl;

  // Check if the GPU is valid
  if (args["device"].as<int>() >= num_gpus) {
    std::cout << "Invalid GPU" << std::endl;
    return;
  }

  printf("Using GPU %d\n", args["device"].as<int>());
  CHECK_CUDA(hipSetDevice(args["device"].as<int>()));

  std::string filename = "";
  if (args.count("market") == 1) {
    filename = args["market"].as<std::string>();
    if (util::is_market(filename)) {
    } else {
      std::cout << options.help({""}) << std::endl;
      std::exit(1);
    }
  } else if (args.count("bin") == 1) {
    filename = args["bin"].as<std::string>();
    if (util::is_binary_csr(filename)) {
    } else {
      std::cout << options.help({""}) << std::endl;
      std::exit(1);
    }
  } else {
    std::cout << options.help({""}) << std::endl;
    std::exit(1);
  }

  // --
  // Define types

  using row_t = int;
  using edge_t = int;
  using nonzero_t = float;
  using csr_t = format::csr_t<memory_space_t::device, row_t, edge_t, nonzero_t>;

  // --
  // IO

  csr_t csr;
  if (util::is_market(filename)) {
    io::matrix_market_t<row_t, edge_t, nonzero_t> mm;
    csr.from_coo(mm.load(filename));
  } else if (util::is_binary_csr(filename)) {
    csr.read_binary(filename);
  } else {
    std::cerr << "Unknown file format: " << filename << std::endl;
    exit(1);
  }

  // Print the GPU stats
  print_gpu_stats(results);

  // Print the matrix stats
  printf("Matrix: %s\n", filename.c_str());
  printf("- Rows: %d\n", csr.number_of_rows);
  printf("- Columns: %d\n", csr.number_of_columns);
  printf("- Nonzeros: %d\n", csr.number_of_nonzeros);
  display(csr.row_offsets, "row_offsets", 40);
  display(csr.column_indices, "column_indices", 40);
  display(csr.nonzero_values, "values", 40);

  results["matrix"]["filename"] = filename.c_str();
  results["matrix"]["rows"] = csr.number_of_rows;
  results["matrix"]["columns"] = csr.number_of_columns;
  results["matrix"]["nonzeros"] = csr.number_of_nonzeros;

  thrust::host_vector<nonzero_t> x_host(csr.number_of_columns);

  srand(0);
  for (size_t idx = 0; idx < x_host.size(); idx++)
    x_host[idx] = rand() % 64;

  // --
  // Init the vectors
  thrust::device_vector<nonzero_t> x_device = x_host;
  thrust::device_vector<nonzero_t> y_device(csr.number_of_rows);

  // --
  // Run the algorithm

  double elapsed_cusparse = 0;
  double elapsed_cub = 0;
  double elapsed_mgpu = 0;
  double elapsed_gunrock = 0;
  double elapsed_tiled = 0;

  if (args.count("cusparse")) {
    elapsed_cusparse =
        test_spmv(CUSPARSE, csr, x_device, y_device, args, results);
  }

  if (args.count("cub")) {
    elapsed_cub = test_spmv(CUB, csr, x_device, y_device, args, results);
  }

  if (args.count("mgpu")) {
    elapsed_mgpu = test_spmv(MGPU, csr, x_device, y_device, args, results);
  }

  if (args.count("gunrock")) {
    elapsed_gunrock =
        test_spmv(GUNROCK, csr, x_device, y_device, args, results);
  }

  if (args.count("tiled")) {
    elapsed_tiled = test_spmv(TILED, csr, x_device, y_device, args, results);
  }

  results["runtime"]["cusparse"] = elapsed_cusparse;
  results["runtime"]["cub"] = elapsed_cub;
  results["runtime"]["mgpu"] = elapsed_mgpu;
  results["runtime"]["gunrock"] = elapsed_gunrock;
  results["runtime"]["tiled"] = elapsed_tiled;

  printf("%s,%d,%d,%d,%d,%f,%f,%f,%f,%f\n", filename.c_str(),
         csr.number_of_rows, csr.number_of_columns, csr.number_of_nonzeros,
         args["pin"].as<bool>(), elapsed_cusparse, elapsed_cub, elapsed_mgpu,
         elapsed_gunrock, elapsed_tiled);

  // Log a success
  results["success"] = true;

  // Save the JSON file
  auto json_filename = args["jsonfile"].as<std::string>();

  if (json_filename == "stdout") {
    std::cout << results.dump(4) << std::endl;
  } else {
    std::ofstream json_file(json_filename);
    json_file << results.dump(4);
    json_file.close();
  }
}

int main(int argc, char** argv) {
  test_spmv(argc, argv);
  return EXIT_SUCCESS;
}
